#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define CHUNK_SIZE 16

struct Arrow {
    uint8_t type;
};

struct Chunk {
    uint16_t x, y;
    Arrow arrows[CHUNK_SIZE * CHUNK_SIZE][2];
};

__global__ void update(Chunk *chunks, unsigned long long steps) {
    for (unsigned long long i = 0; i < steps; i++) {
        // ...
        __syncthreads();
    }
}

/*
thrust::host_vector<Chunk> h_vec;
h_vec.push_back(Chunk { 0, 0 });
h_vec.push_back(Chunk { 10, 0 });
h_vec.push_back(Chunk { 10, 12 });
thrust::device_vector<Chunk> d_vec = h_vec;
Chunk *chunks = thrust::raw_pointer_cast(d_vec.data());
clock_t start = clock();
unsigned long long i = 0;
while ((clock() - start) < 5000) {
    update<<<d_vec.size(), dim3(CHUNK_SIZE, CHUNK_SIZE)>>>(chunks);
    i += 100000;
}
hipDeviceSynchronize();
std::cout << (i / float(clock() - start)) << " iterations per second" << std::endl;
*/