#include "hip/hip_runtime.h"
#include "render.h"

__global__ void render(hipSurfaceObject_t surf, const cudarrows::Chunk *chunks, uint8_t step, int32_t minX, int32_t minY, int32_t maxX, int32_t maxY) {
    cudarrows::Chunk chunk = chunks[blockIdx.x];
    int16_t chunkX = chunk.x;
    int16_t chunkY = chunk.y;
    int32_t x = chunkX * CHUNK_SIZE + threadIdx.x;
    int32_t y = chunkY * CHUNK_SIZE + threadIdx.y;
    if (x < minX || y < minY || x > maxX || y > maxY) return;
    x -= minX;
    y -= minY;
    uint8_t idx = threadIdx.y * CHUNK_SIZE + threadIdx.x;
    cudarrows::Arrow arrow = chunk.arrows[idx];
    cudarrows::ArrowState state = chunk.states[step][idx];
    uchar4 data = { arrow.type, arrow.rotation + 0x4 * arrow.flipped, state.signal, 255 };
    surf2Dwrite(data, surf, x * sizeof(data), y);
}